#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", 
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
upsweep_kernel(int N, int stride, int* input) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * stride * 2;


    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    input[index + stride * 2 - 1] += input[index + stride - 1];
}

__global__ void
downsweep_kernel(int N, int stride, int* input) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = (blockIdx.x * blockDim.x + threadIdx.x) * stride * 2;

    if (stride == 1 && index == N - 1) {
      input[index] = 0;
    }
    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    int t = input[index + stride - 1];
    input[index + stride - 1] = input[index + stride * 2 - 1];
    input[index + stride * 2 - 1] += t;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel segmented scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep input
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    for (int i = 1; i <= N / 2; i*=2) {
      int n_threads = N / (2 * i);
      int blocks = (n_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      upsweep_kernel<<<blocks, THREADS_PER_BLOCK>>>(N, i, input);
      cudaCheckError(hipDeviceSynchronize());
    }

    for (int i = N / 2; i >= 1; i/=2) {
      int n_threads = N / (2 * i);
      int blocks = (n_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
      downsweep_kernel<<<blocks, THREADS_PER_BLOCK>>>(N, i, input);
      cudaCheckError(hipDeviceSynchronize());
    }

    for (int i = 0; i < N; i++) {
      result[i] = input[i];
    }

}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of segmented scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void
check_neighbor_kernel(int length, int* input) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index + 1 < length && input[index] == input[index + 1]) {
      input[index] = 1;
    } else {
      input[index] = 0;
    }
}

__global__ void
get_index_kernel(int length, int* prefix_sum, int* output) {

    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > 0 && prefix_sum[index] == prefix_sum[index - 1] + 1) {
      output[prefix_sum[index] - 1] = index - 1;
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    check_neighbor_kernel<<<blocks, THREADS_PER_BLOCK>>>(length, device_input);
    cudaCheckError(hipDeviceSynchronize());
    int* tmp;
    hipMalloc(&tmp, length * sizeof(int));
    exclusive_scan(device_input, length, tmp);
    get_index_kernel<<<block, THREADS_PER_BLOCK>>>(length, tmp, device_output);
    cudaCheckError(hipDeviceSynchronize());
    return tmp[-1]; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
